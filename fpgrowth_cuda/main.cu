#include "hip/hip_runtime.h"
/*
* author��lzy
   datetime:20160916
*/
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>
#include <map>
#include <vector>
#include "data_struct.cuh"
#include "build_bitMatrix.cuh"
#include "sort_leafnode.cuh"
#include "build_BRTree.cuh"
#include "mine_BRTree.cuh"
/*
main program
TODO: improve algothrim,unit test
analysis algothrim, nsight data analysis
*/
/*
ʾ�����룺
f c a m p
f c a b m
f b
c b p
f c a m p
<min_support_count=3
min_support=0.6
*/
int main(int argc, char** argv)
{
	if (argc < 2){
		cout << "Usage: fpgrowth_gpu <min_support>" << endl;
		return 1;
	}
    //time1:
	cout << "----start----" << endl;
	double min_support = atof(argv[1]);
	int n_trans = 0;
	vector<vector<string>> trans;
	map<string, int> C1;
	//1.��������
	string line;
	while (getline(cin, line)){
		istringstream is(line);
		vector<string> items;
		string item;
		while (is >> item){
			items.push_back(item);
			map<string, int>::iterator it = C1.find(item);
			if (it != C1.end())
				C1[item] += 1;
			else
				C1[item] = 1;
		}
		trans.push_back(items);
		n_trans++;
	}

	//2.����Ƶ��1��������ݼ�λ��
	int n_items = 0;
	map<string, int> item_indexs;
	vector<string> item_names;
	map<string, int>::iterator it = C1.begin();
	int min_support_count = n_trans*min_support;

	for (map<string, int>::iterator it = C1.begin(); it != C1.end(); ++it){
		if (it->second >= min_support_count){
			//cout << it->first << ":" << setprecision(2) << it->second << endl;//���F1�
			item_indexs[it->first] = n_items++;
			item_names.push_back(it->first);
		}
	}

	//3.ת��Ϊλ��
	Prefix_Array* leaf_prefixs = new Prefix_Array(n_trans, n_items);
	trans2Prefix(trans, item_indexs, leaf_prefixs, n_trans);
	//cout << "Ҷ�ӽڵ��ǰ׺����Ϊ��" << endl;
	//display_prefix_array(leaf_prefixs);

	//4.����Ҷ�ӽڵ�
	Node* leaf_node = new Node[n_trans]();
	construct_leaf_node(leaf_node, n_trans, n_items);
	//cout << "��ʼҶ�ӽڵ��ǣ�" << endl;
	//display_leaf_node(leaf_node, n_trans);
	
	//5.��Ҷ�ӽڵ�����ȥ��
	Node *dev_leaf_nodes;
	hipMalloc((void **)&dev_leaf_nodes, sizeof(Node) * n_trans);
	hipMemcpy(dev_leaf_nodes, leaf_node, sizeof(Node) * n_trans, hipMemcpyHostToDevice);

	for (int i = 0; i < n_trans; i += 1){
		parallel_sort_leaf << <n_trans / 512 + 1, 512 >> >(n_trans, n_items, 0, leaf_prefixs->data,
			leaf_prefixs->real_size, dev_leaf_nodes);
		parallel_sort_leaf << <n_trans / 512 + 1, 512 >> >(n_trans, n_items, 1, leaf_prefixs->data,
			leaf_prefixs->real_size, dev_leaf_nodes);

	}
	int *res = 0;
	hipMalloc((void**)&res, sizeof(int));
	parallel_find_size << <n_trans / 512 + 1, 512 >> >(res, dev_leaf_nodes, n_trans);
	hipDeviceSynchronize();
	int new_n_trans = 0;
	hipMemcpy(&new_n_trans, res, sizeof(int),   hipMemcpyDeviceToHost);
	//--------------debug-----��ʼ
	//hipMemcpy(leaf_node, dev_leaf_nodes, sizeof(Node) * n_trans, hipMemcpyDeviceToHost);
	//cout << "������Ҷ�ӽڵ��ǣ�" << endl;
	//display_leaf_node(leaf_node, new_n_trans);
	//---------------debug----���� 
	hipDeviceSynchronize();
	//6.���й������������
	//6-1 �ڲ��ڵ�
	Node *dev_inner_nodes;
	hipMalloc((void **)&dev_inner_nodes, sizeof(Node) * (new_n_trans-1));
	Prefix_Array* inner_prefixs = new Prefix_Array(new_n_trans - 1, n_items);
	//6-2 ͷ��
	HItem *dev_items;
	HItem_Node *dev_item_nodes;
	hipMalloc((void **)&dev_items, sizeof(HItem) * n_items);
	hipMalloc((void **)&dev_item_nodes, sizeof(HItem_Node) * n_items*new_n_trans);

	//6-3.���д��������ڲ��ڵ�
	parallel_build_tree << <(new_n_trans - 1) / 512 + 1, 512 >> >(
		new_n_trans, n_items,
		leaf_prefixs->real_size, leaf_prefixs->data, dev_leaf_nodes,
		inner_prefixs->data, dev_inner_nodes
		);
	hipDeviceSynchronize();
	//--------------debug-----��ʼ
	//Node *inner_node = new Node[new_n_trans-1]();
	//hipMemcpy(leaf_node, dev_leaf_nodes, sizeof(Node) * new_n_trans, hipMemcpyDeviceToHost);
	//hipMemcpy(inner_node, dev_inner_nodes, sizeof(Node) * (new_n_trans-1), hipMemcpyDeviceToHost);
	//cout << "BRTree��Ҷ�ӽڵ���м�ڵ��ǣ�" << endl;
	//display_leaf_node(leaf_node, new_n_trans);
	//cout << "----------" << endl;
	//display_leaf_node(inner_node, new_n_trans - 1);
	//cout << "�м�ڵ��ǰ׺����----------" << endl;
	//display_prefix_array(inner_prefixs);
	//--------------debug-----����
	//6-3.���д���ͷ��
	parallel_build_HItem << <new_n_trans / 512 + 1, 512 >> >(new_n_trans, dev_leaf_nodes,
		leaf_prefixs->data, leaf_prefixs->real_size,
		dev_items, dev_item_nodes, new_n_trans);
	hipDeviceSynchronize();

	parallel_build_HItem << <(new_n_trans - 1) / 512 + 1, 512 >> >(new_n_trans - 1, dev_inner_nodes,
		inner_prefixs->data, inner_prefixs->real_size,
		dev_items, dev_item_nodes, new_n_trans);
	hipDeviceSynchronize();
	//--------------debug-----��ʼ
	//HItem *items = new HItem[n_items];
	//HItem_Node *item_nodes = new HItem_Node[n_items*new_n_trans];
	//hipMemcpy(items, dev_items, sizeof(HItem) * n_items, hipMemcpyDeviceToHost);
	//hipMemcpy(item_nodes, dev_item_nodes, sizeof(HItem_Node) * n_items*new_n_trans, hipMemcpyDeviceToHost);
	//cout << "-------------" << endl;
	//display_HItem(items, n_items);
	//cout << "ͷ��ڵ�-------------" << endl;
	//display_HItem_Node(item_nodes, n_items, new_n_trans);
	//--------------debug-----����

	//7.�����ھ���������
	FreqItems nullitems;
	nullitems.k = 0;
	unsigned int res_size = pow(2, n_items);
	FreqItems * dev_results;
	hipMalloc((void **)&dev_results, sizeof(FreqItems)* res_size);
	/*int *z=0;
	hipMalloc((void**)&z, sizeof(int));
	hipMemset(z, 0, 1);*/
	//cout << new_n_trans << " " << n_items << " " << min_support_count << endl;
	parelle_mine_fptree << <n_items / 512 + 1, 512 >> >
		(new_n_trans, n_items, min_support_count,
		leaf_prefixs->data, inner_prefixs->data, n_items / 32 + 1,
		nullitems, dev_items, dev_item_nodes,
		dev_leaf_nodes, dev_inner_nodes,
		dev_results);
	hipDeviceSynchronize();
	
	//8.������
	FreqItems * results = new FreqItems[res_size]() ;
	hipMemcpy(results, dev_results, sizeof(FreqItems) *res_size, hipMemcpyDeviceToHost);
	display_freq(results, res_size);

	hipFree(dev_leaf_nodes);
	hipFree(dev_inner_nodes);
	hipFree(dev_items);
	hipFree(dev_item_nodes);
	hipFree(dev_results);
	delete leaf_prefixs;
	delete inner_prefixs;
	delete results;
	/*
	ע���ڴ�й¶����TODO:����ڴ�й¶����
	*/
	cout << "----end----" << endl;
	system("pause");
    return 0;
}


